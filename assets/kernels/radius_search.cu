#include "hip/hip_runtime.h"
// Fast Radius Search Exploiting Ray Tracing Frameworks
// Authors: I. Evangelou, G. Papaioannou, K. Vardis, A. A. Vasilakis

#pragma once

#include <optix.h>
#include <stdint.h>
#include <cstdio>

#include "../../source/sutil/vec_math.hpp"
#include "../../source/cuda_types.hpp"

namespace bvh_radSearch
{
    struct payload_t
    {
        query_t query;
        uint32_t count;

        uint32_t offset;
        int32_t maxDistElemi;
        int32_t foundNeighbors;
        float_t maxDistElemf;
    };

    __device__ __forceinline__ void* unpackPointer(
        uint32_t i0,
        uint32_t i1) noexcept
    {
        const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
        void* ptr = reinterpret_cast<void*>(uptr);
        return ptr;
    }

    __device__ __forceinline__ void packPointer(
        const void* ptr,
        uint32_t& i0,
        uint32_t& i1) noexcept
    {
        const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
        i0 = uptr >> 32;
        i1 = uptr & 0x00000000ffffffff;
    }

    template<typename T>
    __device__ __forceinline__ T* getPayload(void) noexcept
    {
        const uint32_t u0 = optixGetPayload_0();
        const uint32_t u1 = optixGetPayload_1();
        return reinterpret_cast<T*>(unpackPointer(u0, u1));
    }

    extern "C" { __constant__ Params params; }

    __device__ void findLargestDist(payload_t& payload) noexcept
    {
        payload.maxDistElemi = payload.offset;
        payload.maxDistElemf = params.optixDists[payload.maxDistElemi];

        for (int32_t k = 1; k < params.knn; ++k)
        {
            float_t tmpDist = params.optixDists[payload.offset + k];
            if (tmpDist > payload.maxDistElemf)
            {
                payload.maxDistElemi = payload.offset + k;
                payload.maxDistElemf = tmpDist;
            }
        }
    }

    extern "C" __global__ void __raygen__radSearch_count_bruteforce(void)
    {
        const uint3& idx = optixGetLaunchIndex();
        query_t& query = params.queries[idx.x];
        query.count = 0;

        for (size_t s = 0; s < params.numSamples; ++s)
        {
            const float3 diff = params.samplePos[s] - query.position;
            const float_t t = dot(diff, diff);

            if (t < query.radius * query.radius)
            {
                ++query.count;
            }
        }

        atomicAdd(&params.totalCount[0], query.count);
        atomicMax(&params.maxCount[0], query.count);
        atomicMin(&params.minCount[0], query.count);
    }

    extern "C" __global__ void __intersection__radSearch_count_bruteforce(void) { /* Empty */ }

    extern "C" __global__ void __raygen__radSearch_count(void)
    {
        const uint3 & idx = optixGetLaunchIndex();
        query_t & query = params.queries[idx.x];
        payload_t payload;
        payload.query = query;
        payload.count = 0;

        uint32_t u0, u1;
        packPointer(&payload, u0, u1);

        optixTrace(params.gasHandle,
            query.position, make_float3(1.e-16f),
            0.f, 1.e-16f, 0.f,
            OptixVisibilityMask(1),
            OPTIX_RAY_FLAG_DISABLE_ANYHIT |
            OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
            0, 4, 0,
            u0, u1);

        query.count = payload.count;
        atomicAdd(&params.totalCount[0], payload.count);
        atomicMax(&params.maxCount[0], payload.count);
        atomicMin(&params.minCount[0], payload.count);
    }

    extern "C" __global__ void __intersection__radSearch_count(void)
    {
        payload_t& payload = *getPayload<payload_t>();

        float3& sample = params.samplePos[optixGetPrimitiveIndex()];

        const float3 diff = sample - optixGetWorldRayOrigin();
        const float_t t = dot(diff, diff);

        if (t < payload.query.radius * payload.query.radius)
        {
            ++payload.count;
        }
    }

    extern "C" __global__ void __raygen__radSearch(void)
    {
        const uint3& idx = optixGetLaunchIndex();
        query_t& query = params.queries[idx.x];
        payload_t payload;
        payload.query = query;
        payload.count = 0;
        payload.offset = idx.x * params.knn;
        payload.maxDistElemi = idx.x * params.knn;
        payload.maxDistElemf = query.radius + 1.f;
        payload.foundNeighbors = 0;
        query.count = params.knn;

        uint32_t u0, u1;
        packPointer(&payload, u0, u1);

        optixTrace(params.gasHandle,
            query.position, make_float3(1.e-16f),
            0.f, 1.e-16f, 0.f,
            OptixVisibilityMask(1),
            OPTIX_RAY_FLAG_DISABLE_ANYHIT |
            OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
            0, 4, 0,
            u0, u1);
    }

    extern "C" __global__ void __intersection__radSearch(void)
    {
        payload_t& payload = *getPayload<payload_t>();

        float3& sample = params.samplePos[optixGetPrimitiveIndex()];

        const float3 diff = sample - optixGetWorldRayOrigin();
        const float_t t = dot(diff, diff);

        if (t < payload.query.radius * payload.query.radius)
        {
            if (t < payload.maxDistElemf)
            {
                if (payload.foundNeighbors < params.knn)
                {
                    const uint32_t idxToSave = payload.offset + payload.foundNeighbors;
                    params.optixIndices[idxToSave] = optixGetPrimitiveIndex();
                    params.optixDists[idxToSave] = t;

                    if (payload.foundNeighbors == params.knn - 1)
                    {
                        findLargestDist(payload);
                    }

                    ++payload.foundNeighbors;
                }
                else
                {
                    params.optixIndices[payload.maxDistElemi] = optixGetPrimitiveIndex();
                    params.optixDists[payload.maxDistElemi] = t;
                    findLargestDist(payload);
                }
            }
        }
    }

    extern "C" __global__ void __raygen__radSearch_bruteforce(void)
    {
        const uint3& idx = optixGetLaunchIndex();
        query_t& query = params.queries[idx.x];
        payload_t payload;
        payload.query = query;
        payload.count = 0;
        payload.offset = idx.x * params.knn;
        payload.maxDistElemi = idx.x * params.knn;
        payload.maxDistElemf = query.radius + 1.f;
        payload.foundNeighbors = 0;
        query.count = params.knn;

        uint32_t u0, u1;
        packPointer(&payload, u0, u1);

        for (size_t s = 0; s < params.numSamples; ++s)
        {
            const float3 diff = params.samplePos[s] - query.position;
            const float_t t = dot(diff, diff);

            if (t < query.radius * query.radius)
            {
                if (t < payload.maxDistElemf)
                {
                    if (payload.foundNeighbors < params.knn)
                    {
                        const uint32_t idxToSave = payload.offset + payload.foundNeighbors;
                        params.optixIndices[idxToSave] = s;
                        params.optixDists[idxToSave] = t;

                        if (payload.foundNeighbors == params.knn - 1)
                        {
                            findLargestDist(payload);
                        }

                        ++payload.foundNeighbors;
                    }
                    else
                    {
                        params.optixIndices[payload.maxDistElemi] = s;
                        params.optixDists[payload.maxDistElemi] = t;
                        findLargestDist(payload);
                    }
                }
            }
        }
    }

    extern "C" __global__ void __intersection__radSearch_bruteforce(void) { /* Empty */ }
}